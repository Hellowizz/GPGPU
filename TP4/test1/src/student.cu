#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{

	__global__ void greyCUDA(const int width, const int height, 
		const uchar *const dev_input, uchar *const dev_output)
	{
			// id global en x
		const int idThreadGX = threadIdx.x // id du thread dans le block 
							+ blockIdx.x  // id du block dans la grid
							* blockDim.x;  // taille d'un block, nb threads dans blocks
			// nb threads global en x
		const int nbThreadsGX = blockDim.x 
							* gridDim.x; // nb blocks dans grid

							// id global en y
		const int idThreadGY = threadIdx.y // id du thread dans le block 
							+ blockIdx.y  // id du block dans la grid
							* blockDim.y;  // taille d'un block, nb threads dans blocks
			// nb threads global en y
		const int nbThreadsGY = blockDim.y 
							* gridDim.y; // nb blocks dans grid

		for (int idY = idThreadGY; idY < height; idY += nbThreadsGY)
		{
			for(int idX = idThreadGX; idX < width; idX += nbThreadsGX){
				int id = (idY * width + idX) * 3;

				uchar greyVal = fminf(255.f, 0.299f * dev_input[id] + 0.587f * dev_input[id+1] + 0.114f * dev_input[id+2]);
				dev_output[id] = greyVal;
				dev_output[id+1] = greyVal;
				dev_output[id+2] = greyVal;
			}
		}	
	}

	__global__ void rgbTOhsvCUDA(const uchar *const dev_input, const int width, const int height, 
		float *const dev_outputH, float *const dev_outputS, float *const dev_outputV)
	{
			// id global en x
		const int idThreadGX = threadIdx.x // id du thread dans le block 
							+ blockIdx.x  // id du block dans la grid
							* blockDim.x;  // taille d'un block, nb threads dans blocks
			// nb threads global en x
		const int nbThreadsGX = blockDim.x 
							* gridDim.x; // nb blocks dans grid

							// id global en y
		const int idThreadGY = threadIdx.y // id du thread dans le block 
							+ blockIdx.y  // id du block dans la grid
							* blockDim.y;  // taille d'un block, nb threads dans blocks
			// nb threads global en y
		const int nbThreadsGY = blockDim.y 
							* gridDim.y; // nb blocks dans grid

		double min, max, delta;

		for (int idY = idThreadGY; idY < height; idY += nbThreadsGY)
		{
			for(int idX = idThreadGX; idX < width; idX += nbThreadsGX){

				const uint idInRGB = (idY * width + idX) * 3;
				const uint idInHSV = (idY * width + idX);

				const uchar inR = dev_input[idInRGB];
				const uchar inG = dev_input[idInRGB + 1];
				const uchar inB = dev_input[idInRGB + 2];

				min = inR < inG ? inR : inG;
		    	min = min  < inB ? min : inB;

		    	max = inR > inG ? inR : inG;
		    	max = max  > inB ? max  : inB;

		    	dev_outputV[idInHSV] = max;

		    	delta = max - min;
		    	if (delta < 0.00001)
			    {
			        dev_outputS[idInHSV] = 0;
			        dev_outputH[idInHSV] = 0;
			        continue;
			    }
			    if (max > 0.0)
			    {
			    	dev_outputS[idInHSV] = (delta/ max);
			    } else {
			    	dev_outputS[idInHSV] = 0.0;
			        dev_outputH[idInHSV] = NAN;                           // its now undefined
			        continue;
			    }

				if( inR >= max )                           // > is bogus, just keeps compilor happy
			        dev_outputH[idInHSV] = ( inG - inB ) / delta;        // between yellow & magenta
			    else if( inG >= max )
			        dev_outputH[idInHSV] = 2.0 + ( inB - inR ) / delta;  // between cyan & yellow
			    else
			        dev_outputH[idInHSV] = 4.0 + ( inR - inG ) / delta;  // between magenta & cyan

			    dev_outputH[idInHSV] *= 60.0;                              // degrees

			    if( dev_outputH[idInHSV] < 0.0 )
			        dev_outputH[idInHSV] += 360.0;
			}
		}	
	}

	__global__ void hsvTOrgbCUDA(const float *const dev_inputH, const float *const dev_inputS, const float *const dev_inputV,
		const int width, const int height, 
		uchar *const dev_outputRGB)
	{
			// id global en x
		const int idThreadGX = threadIdx.x // id du thread dans le block 
							+ blockIdx.x  // id du block dans la grid
							* blockDim.x;  // taille d'un block, nb threads dans blocks
			// nb threads global en x
		const int nbThreadsGX = blockDim.x 
							* gridDim.x; // nb blocks dans grid

							// id global en y
		const int idThreadGY = threadIdx.y // id du thread dans le block 
							+ blockIdx.y  // id du block dans la grid
							* blockDim.y;  // taille d'un block, nb threads dans blocks
			// nb threads global en y
		const int nbThreadsGY = blockDim.y 
							* gridDim.y; // nb blocks dans grid

		double hh, p, q, t, ff;
	    long k;

		for (int idY = idThreadGY; idY < height; idY += nbThreadsGY)
		{
			for(int idX = idThreadGX; idX < width; idX += nbThreadsGX){
				const uint idInRGB = (idY * width + idX) * 3;
				const uint idInHSV = (idY * width + idX);

				const uint idOutR = idInRGB;
				const uint idOutG = idInRGB + 1;
				const uint idOutB = idInRGB + 2;

				hh = dev_inputH[idInHSV];

			    if(hh >= 360.0) hh = 0.0;
			    hh /= 60.0;
			    k = (long)hh;
			    ff = hh - k;
			    p = dev_inputV[idInHSV] * (1.0 - dev_inputS[idInHSV]);
			    q = dev_inputV[idInHSV] * (1.0 - (dev_inputS[idInHSV] * ff));
			    t = dev_inputV[idInHSV] * (1.0 - (dev_inputS[idInHSV] * (1.0 - ff)));

			    switch(k) {
			    case 0:
			        dev_outputRGB[idOutR] = dev_inputV[idInHSV];
			        dev_outputRGB[idOutG] = t;
			        dev_outputRGB[idOutB] = p;
			        break;
			    case 1:
			        dev_outputRGB[idOutR] = q;
			        dev_outputRGB[idOutG] = dev_inputV[idInHSV];
			        dev_outputRGB[idOutB] = p;
			        break;
			    case 2:
			        dev_outputRGB[idOutR] = p;
			        dev_outputRGB[idOutG] = dev_inputV[idInHSV];
			        dev_outputRGB[idOutB] = t;
			        break;
			    case 3:
			        dev_outputRGB[idOutR] = p;
			        dev_outputRGB[idOutG] = q;
			        dev_outputRGB[idOutB] = dev_inputV[idInHSV];
			        break;
			    case 4:
			        dev_outputRGB[idOutR] = t;
			        dev_outputRGB[idOutG] = p;
			        dev_outputRGB[idOutB] = dev_inputV[idInHSV];
			        break;
			    case 5:
			    default:
			        dev_outputRGB[idOutR] = dev_inputV[idInHSV];
			        dev_outputRGB[idOutG] = p;
			        dev_outputRGB[idOutB] = q;
			        break;
			    }
			}
		}
	}

	void studentJob(const std::vector<uchar> &input, const uint width, const uint height, std::vector<uchar> &output)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar *dev_input = NULL;
		float *dev_outputH = NULL;
		float *dev_outputS = NULL;
		float *dev_outputV = NULL;
		uchar *dev_output = NULL;
		uchar *dev_histo = NULL;
		
		std::cout 	<< "Allocating 2 arrays: ";
		chrGPU.start();
		const size_t bytes = input.size() * sizeof(uchar);
		const size_t HSVbytes = input.size() * sizeof(float);
		
		hipMalloc((void **) &dev_input, bytes);
		hipMalloc((void **) &dev_outputH, HSVbytes);
		hipMalloc((void **) &dev_outputS, HSVbytes);
		hipMalloc((void **) &dev_outputV, HSVbytes);
		hipMalloc((void **) &dev_output, bytes);
		hipMalloc((void **) &dev_histo, 255 * sizeof(uchar));

		chrGPU.stop();
		std::cout 	<< "Allocation -> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		std::cout 	<< "Copying data to GPU : ";
		chrGPU.start();
		// Copy data from host to device (input arrays) 
		hipMemcpy(dev_input, input.data(), bytes, hipMemcpyHostToDevice);
		chrGPU.stop();
		std::cout 	<< "Copying -> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Launch the kernel for the greylvl image
		chrGPU.start();//dim3
		std::cout 	<< "Lauching the kernel";
		rgbTOhsvCUDA<<<dim3(16, 16), dim3(32, 32)>>>(dev_input, width, height, dev_outputH, dev_outputS, dev_outputV);
		hsvTOrgbCUDA<<<dim3(16, 16), dim3(32, 32)>>>(dev_outputH, dev_outputS, dev_outputV, width, height, dev_output);
		chrGPU.stop();
		std::cout 	<< "Calculations -> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		std::cout 	<< "Copying data to CPU : ";
		chrGPU.start();
		// Copy data from device to host (output array)  
		hipMemcpy(output.data(), dev_output, bytes, hipMemcpyDeviceToHost);
		chrGPU.stop();
		std::cout 	<< "Copying -> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		hipFree(dev_input);
		hipFree(dev_outputH);
		hipFree(dev_outputS);
		hipFree(dev_outputV);
		hipFree(dev_output);
		hipFree(dev_histo);
	}
}
